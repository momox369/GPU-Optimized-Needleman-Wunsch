#include "hip/hip_runtime.h"

#include <assert.h>

#include "common.h"
#include "timer.h"

__global__ void kernel_nw2_(unsigned char* sequence1, unsigned char* sequence2, int* scores_d, unsigned int numSequences)
{
    __shared__ int previousDiagonal[SEQUENCE_LENGTH];
    __shared__ int sm_sequence2[SEQUENCE_LENGTH];    

    // 2 - Compute the scores for the rest of the matrix
    int threadIteration = 1;
    int count = 2 * SEQUENCE_LENGTH - 1;
    int col = threadIdx.x+1;
    int top     = (col)  * DELETION;
    int topLeft = threadIdx.x * DELETION;
    //if not 1st row and not 1st col, take the value diiagonally above and to the left

    // sm_sequence1[threadIdx.x] = sequence1[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    sm_sequence2[threadIdx.x] = sequence2[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    int seq1Value = sequence1[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    //  __syncthreads();

    int row = threadIteration; // the row being addressed, starts at 2
    for (unsigned int diagIndex = 0 ; diagIndex <= count ; ++diagIndex) {
        // Compute current diagonal from left to right bottom to top
           
	    if (threadIdx.x <= diagIndex && row <= SEQUENCE_LENGTH && col <= SEQUENCE_LENGTH) {
            int left    = (col == 1) ? (row)  * INSERTION : (previousDiagonal[col-2]); //else, take the value directly to the left of it
            int insertion = top + INSERTION;
            int deletion  = left + DELETION;
            int match     = topLeft + (
                // (sequence2[blockIdx.x*SEQUENCE_LENGTH + (row-1)] == sequence1[blockIdx.x*SEQUENCE_LENGTH + (col-1)])
                (sm_sequence2[(row-1)] == seq1Value)
                ? MATCH 
                : MISMATCH
                ); //check if there is a match
            int max = (insertion > deletion) ? insertion : deletion; 
            max = (match > max) ? match : max;
            // currentDiagonal[threadIdx.x] = max; //store it in the matrix
            top = max;
            topLeft = left;
            ++row;
        }

        __syncthreads(); 
        previousDiagonal[threadIdx.x] = top;
	     __syncthreads();
	}


    if(threadIdx.x == blockDim.x - 1){
      //printf("-- %d -- ** %d ** \n",previousDiagonal[SEQUENCE_LENGTH-1],previousDiagonal[SEQUENCE_LENGTH-2]);
      // 3 - Write the final score to the output array
      scores_d[blockIdx.x] = top;
    }
}

__global__ void kernel_nw2(unsigned char* sequence1, unsigned char* sequence2, int* scores_d, unsigned int numSequences)
{
    __shared__ int previousDiagonal[SEQUENCE_LENGTH];
    __shared__ int sm_sequence2[SEQUENCE_LENGTH];    

    int count   = 2 * SEQUENCE_LENGTH - 1;
    // 2 - Compute the scores for the rest of the matrix
    int row     = 0;
    int col     = threadIdx.x;
    int top     = (col+1)  * DELETION;
    int topLeft = (col) * DELETION;

    // sm_sequence1[threadIdx.x] = sequence1[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    sm_sequence2[threadIdx.x] = sequence2[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    int seq1Value = sequence1[blockIdx.x*SEQUENCE_LENGTH + col];
    //  __syncthreads();

    for (unsigned int diagIndex = 0 ; diagIndex <= count ; ++diagIndex) {
        // Compute current diagonal from left to right bottom to top
           
	    if (threadIdx.x <= diagIndex && row < SEQUENCE_LENGTH && col < SEQUENCE_LENGTH) {
            int left    = (col == 0) ? (row+1)  * INSERTION : (previousDiagonal[col-1]); //else, take the value directly to the left of it
            int insertion = top + INSERTION;
            int deletion  = left + DELETION;
            int match     = topLeft + (
                // (sequence2[blockIdx.x*SEQUENCE_LENGTH + (row-1)] == sequence1[blockIdx.x*SEQUENCE_LENGTH + (col-1)])
                (sm_sequence2[(row)] == seq1Value)
                ? MATCH 
                : MISMATCH
                ); //check if there is a match
            int max = (insertion > deletion) ? insertion : deletion; 
            max = (match > max) ? match : max;
            // currentDiagonal[threadIdx.x] = max; //store it in the matrix
            top = max;
            topLeft = left;
            ++row;
        }

        __syncthreads(); 
        previousDiagonal[threadIdx.x] = top;
	     __syncthreads();
	}


    if(threadIdx.x == blockDim.x - 1){
      //printf("-- %d -- ** %d ** \n",previousDiagonal[SEQUENCE_LENGTH-1],previousDiagonal[SEQUENCE_LENGTH-2]);
      // 3 - Write the final score to the output array
      scores_d[blockIdx.x] = top;
    }
}


void nw_gpu2(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {

    assert(SEQUENCE_LENGTH <= 1024); // You can assume the sequence length is not more than 1024

    const unsigned int numThreadsPerBlock = SEQUENCE_LENGTH;
    const unsigned int numBlocks = numSequences;
    //Launching the kernel
    hipDeviceSynchronize();
    kernel_nw2 <<< numBlocks, numThreadsPerBlock >>> (sequence1_d, sequence2_d, scores_d, numSequences);

}
