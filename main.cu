#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <unistd.h>

#include "common.h"
#include "timer.h"

void nw_cpu(unsigned char* sequence1, unsigned char* sequence2, int* scores, unsigned int numSequences) {
    for(unsigned int s = 0; s < numSequences; ++s) {
        //We are assuming that both sequences are of the same length
        int matrix[SEQUENCE_LENGTH][SEQUENCE_LENGTH]; //including the gaps

        //looping over each element of the matrix
        for(int i2 = 0; i2 < SEQUENCE_LENGTH; ++i2) { //rows
            for (int i1 = 0; i1 < SEQUENCE_LENGTH; ++i1) { //cols 

                // Get neighbors
                int top     = (i2 == 0)? ((i1 + 1)*DELETION) : //if first row, initialize to n*deletion
                              (matrix[i2 - 1][i1]); //else, take the value directly above it

                int left    = (i1 == 0)? ((i2 + 1)*INSERTION) : //if first column, initialize to n*insertion
                              (matrix[i2][i1 - 1]); //else, take the value directly to the left of it

                int topleft = (i2 == 0)? (i1*DELETION): //if first row, initialize to col*deletion, else:
                              ((i1 == 0)?(i2*INSERTION): //if not 1st row but it is 1st col, initialize to row*insertion
                                  (matrix[i2 - 1][i1 - 1])); //if not 1st row and not 1st col, take the value diagonally above and to the left

                // Find scores based on neighbors
                int insertion = top + INSERTION;
                int deletion  = left + DELETION;
                int match     = topleft + 
                                ((sequence2[s*SEQUENCE_LENGTH + i2] == sequence1[s*SEQUENCE_LENGTH + i1]) ? MATCH : MISMATCH); //check if there is a match

                // Select best score
                //get max(insertion, deletion, match)
                int max = (insertion > deletion) ? insertion : deletion; 
                max = (match > max)?match:max;
                matrix[i2][i1] = max; //store it in the matrix
            }
        }
        //Final score of the sequence S is the value at the bottom right of the matrix
        //later: back-track based on the max value
        scores[s] = matrix[SEQUENCE_LENGTH - 1][SEQUENCE_LENGTH - 1];
    }
}

void verify(int* scores_cpu, int* scores_gpu, unsigned int numSequences) {
    for(unsigned int s = 0; s < numSequences; ++s) {
        if(scores_cpu[s] != scores_gpu[s]) {
            printf("\033[1;31mMismatch at sequence s = %u (CPU result = %d, GPU result = %d)\033[0m\n", s, scores_cpu[s], scores_gpu[s]);
            return;
        } else { printf("score = %d\n", scores_cpu[s]); // XXX
        }
    }
    printf("Verification succeeded\n");
}

void mutateSequence(unsigned char* sequence1, unsigned char* sequence2) {
    const float PROB_MATCH = 0.80f;
    const float PROB_INS   = 0.10f;
    const float PROB_DEL   = 1.00f - PROB_MATCH - PROB_INS;
    assert(PROB_MATCH >= 0.00f && PROB_MATCH <= 1.00f);
    assert(PROB_INS   >= 0.00f && PROB_INS   <= 1.00f);
    assert(PROB_DEL   >= 0.00f && PROB_DEL   <= 1.00f);
    unsigned int i1 = 0, i2 = 0;
    while(i1 < SEQUENCE_LENGTH && i2 < SEQUENCE_LENGTH) {
        float prob = rand()*1.0f/RAND_MAX;
        if(prob < PROB_MATCH) {
            sequence2[i2++] = sequence1[i1++]; // Match
        } else if(prob < PROB_MATCH + PROB_INS) {
            sequence2[i2++] = rand()%256; // Insertion
        } else {
            ++i1; // Deletion
        }
    }
    while(i2 < SEQUENCE_LENGTH) {
        sequence2[i2++] = rand()%256; // Tail insertions
    }
}

int main(int argc, char**argv) {

    hipDeviceSynchronize();

    // Parse arguments
    unsigned int numSequences = 3000;
    unsigned int runGPUVersion0 = 0;
    unsigned int runGPUVersion1 = 0;
    unsigned int runGPUVersion2 = 0;
    unsigned int runGPUVersion3 = 0;
    int opt;
    while((opt = getopt(argc, argv, "N:0123")) >= 0) {
        switch(opt) {
            case 'N': numSequences = atoi(optarg);  break;
            case '0': runGPUVersion0 = 1;           break;
            case '1': runGPUVersion1 = 1;           break;
            case '2': runGPUVersion2 = 1;           break;
            case '3': runGPUVersion3 = 1;           break;
            default:  fprintf(stderr, "\nUnrecognized option!\n");
                      exit(0);
        }
    }

    // Allocate memory and initialize data
    printf("Initializing %u sequence pairs of length %u per pair\n", numSequences, SEQUENCE_LENGTH);
    Timer timer;
    unsigned char* sequence1 = (unsigned char*) malloc(numSequences*SEQUENCE_LENGTH*sizeof(unsigned char));
    unsigned char* sequence2 = (unsigned char*) malloc(numSequences*SEQUENCE_LENGTH*sizeof(unsigned char));
    int* scores_cpu = (int*) malloc(numSequences*sizeof(int));
    int* scores_gpu = (int*) malloc(numSequences*sizeof(int));
    for(unsigned int s = 0; s < numSequences; ++s) {
        for(unsigned int i = 0; i < SEQUENCE_LENGTH; ++i) {
            sequence1[s*SEQUENCE_LENGTH + i] = rand()%256;
        }
        mutateSequence(&sequence1[s*SEQUENCE_LENGTH], &sequence2[s*SEQUENCE_LENGTH]);
    }

    // Compute on CPU
    startTime(&timer);
    nw_cpu(sequence1, sequence2, scores_cpu, numSequences);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time", CYAN);

    if(runGPUVersion0 || runGPUVersion1 || runGPUVersion2 || runGPUVersion3) {

        // Allocate GPU memory
        startTime(&timer);
        unsigned char *sequence1_d;
        unsigned char *sequence2_d;
        int *scores_d;
        hipMalloc((void**) &sequence1_d, numSequences*SEQUENCE_LENGTH*sizeof(unsigned char));
        hipMalloc((void**) &sequence2_d, numSequences*SEQUENCE_LENGTH*sizeof(unsigned char));
        hipMalloc((void**) &scores_d, numSequences*sizeof(int));
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Allocation time");

        // Copy data to GPU
        startTime(&timer);
        hipMemcpy(sequence1_d, sequence1, numSequences*SEQUENCE_LENGTH*sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpy(sequence2_d, sequence2, numSequences*SEQUENCE_LENGTH*sizeof(unsigned char), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Copy to GPU time");

        if(runGPUVersion0) {

            // Reset
            hipMemset(scores_d, 0, numSequences*sizeof(int));
            hipDeviceSynchronize();

            // Compute on GPU with version 0
            startTime(&timer);
            nw_gpu0(sequence1_d, sequence2_d, scores_d, numSequences);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "GPU kernel time (version 0)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(scores_gpu, scores_d, numSequences*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(scores_cpu, scores_gpu, numSequences);

        }

        if(runGPUVersion1) {

            // Reset
            hipMemset(scores_d, 0, numSequences*sizeof(int));
            hipDeviceSynchronize();

            // Compute on GPU with version 1
            startTime(&timer);
            nw_gpu1(sequence1_d, sequence2_d, scores_d, numSequences);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "GPU kernel time (version 1)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(scores_gpu, scores_d, numSequences*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(scores_cpu, scores_gpu, numSequences);

        }

        if(runGPUVersion2) {

            // Reset
            hipMemset(scores_d, 0, numSequences*sizeof(int));
            hipDeviceSynchronize();

            // Compute on GPU with version 2
            startTime(&timer);
            nw_gpu2(sequence1_d, sequence2_d, scores_d, numSequences);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "GPU kernel time (version 2)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(scores_gpu, scores_d, numSequences*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(scores_cpu, scores_gpu, numSequences);

        }

        if(runGPUVersion3) {


            // Reset
            hipMemset(scores_d, 0, numSequences*sizeof(int));
            hipDeviceSynchronize();

            // Compute on GPU with version 3
            startTime(&timer);
            nw_gpu3(sequence1_d, sequence2_d, scores_d, numSequences);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "GPU kernel time (version 3)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(scores_gpu, scores_d, numSequences*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(scores_cpu, scores_gpu, numSequences);

        }

        // Free GPU memory
        startTime(&timer);
        hipFree(sequence1_d);
        hipFree(sequence2_d);
        hipFree(scores_d);
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Deallocation time");

    }

    // Free memory
    free(sequence1);
    free(sequence2);
    free(scores_cpu);
    free(scores_gpu);

    return 0;

}

