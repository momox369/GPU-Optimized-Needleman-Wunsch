#include "hip/hip_runtime.h"

#include <assert.h>
#include <math.h>

#include "common.h"
#include "timer.h"


// shared memory
__global__ void kernel_nw2(unsigned char* sequence1, unsigned char* sequence2, int* scores_d, unsigned int numSequences)
{
    __shared__ int currentDiagonal[SEQUENCE_LENGTH];
    __shared__ int previousDiagonal[SEQUENCE_LENGTH];
    __shared__ int previousPreviousDiagonal[SEQUENCE_LENGTH];
    
    __shared__ int sm_sequence1[SEQUENCE_LENGTH];    
    __shared__ int sm_sequence2[SEQUENCE_LENGTH];    

    // 2 - Compute the scores for the rest of the matrix
    int threadIteration = 1;
    int diagonalLen = sqrt( (float)2) * SEQUENCE_LENGTH;
    int count = 2 * SEQUENCE_LENGTH - 1;
    int col = threadIdx.x+1;
    
    sm_sequence1[threadIdx.x] = sequence1[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
    sm_sequence2[threadIdx.x] = sequence2[blockIdx.x*SEQUENCE_LENGTH + threadIdx.x];
     __syncthreads();

    // TODO: Optimize the memory access pattern
    // seqMatrix[0] = 0;
    // seqMatrix[threadIdx.x + 1] = (threadIdx.x + 1) * DELETION;
    // seqMatrix[(threadIdx.x + 1) * matrixDim] = (threadIdx.x + 1) * INSERTION;

    int currentTop  = (col) * DELETION;     // previousDiagonal[col-1], 
    //     left1       = INSERTION,            // previousDiagonal[col-2]
    //     topLeft1    = ;   // previousPreviousDiagonal[col-2]
    // int count = diagonalLen + col + 100;

    __syncthreads();

    for (unsigned int diagIndex = 0 ; diagIndex <= count ; ++diagIndex) {
        // Compute current diagonal from left to right bottom to top
        int row = threadIteration; // the row being addressed, starts at 2
           
	    if (threadIdx.x <= diagIndex && row <= SEQUENCE_LENGTH && col <= SEQUENCE_LENGTH) {
            ++threadIteration;
            int top     = (row == 1) ? (col)  * DELETION  : (previousDiagonal[col-1]); //else, take the value directly above it
            int left    = (col == 1) ? (row)  * INSERTION : (previousDiagonal[col-2]); //else, take the value directly to the left of it
            int topleft = (row == 1) ? (col-1)* DELETION  : (col == 1)? (row-1)*INSERTION : previousPreviousDiagonal[col-2]; //if not 1st row and not 1st col, take the value diiagonally above and to the left
            int insertion = top + INSERTION;
            int deletion  = left + DELETION;
            int match     = topleft + (
                // (sequence2[blockIdx.x*SEQUENCE_LENGTH + (row-1)] == sequence1[blockIdx.x*SEQUENCE_LENGTH + (col-1)])
                (sm_sequence2[(row-1)] == sm_sequence1[(col-1)])
                ? MATCH 
                : MISMATCH
                ); //check if there is a match
            int max = (insertion > deletion) ? insertion : deletion; 
            max = (match > max) ? match : max;
            currentDiagonal[col-1] = max; //store it in the matrix

            // if(row > 1) {
            //     top     = (row == 1) ? (col)  * DELETION  : (previousDiagonal[col-1]); //else, take the value directly above it
            //     left    = (col == 1) ? (row)  * INSERTION : (previousDiagonal[col-2]); //else, take the value directly to the left of it
            //     topleft = (row == 1) ? (col-1)* DELETION  : (col == 1)? (row-1)*INSERTION : previousPreviousDiagonal[col-2]; //if not 1st row and not 1st col, take the value diiagonally above and to the left
            //     insertion = top + INSERTION;
            //     deletion  = left + DELETION;
            //     match     = topleft + (
            //         (sequence2[blockIdx.x*SEQUENCE_LENGTH + (row-1)] == sequence1[blockIdx.x*SEQUENCE_LENGTH + (col-1)])
            //         ? MATCH 
            //         : MISMATCH
            //         ); //check if there is a match
            //     max = (insertion > deletion) ? insertion : deletion; 
            //     max = (match > max) ? match : max;
            //     currentDiagonal[col-1] = max; //store it in the matrix
            // }
        }
        __syncthreads(); 

        previousPreviousDiagonal[col-1] = previousDiagonal[col-1];
        previousDiagonal[col-1] = currentDiagonal[col-1];

	     __syncthreads();
	}


    if(threadIdx.x == 0){
      //printf("-- %d -- ** %d ** \n",previousDiagonal[SEQUENCE_LENGTH-1],previousDiagonal[SEQUENCE_LENGTH-2]);
      // 3 - Write the final score to the output array
      scores_d[blockIdx.x] = currentDiagonal[SEQUENCE_LENGTH-1];
    }
}



void nw_gpu2(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {

    assert(SEQUENCE_LENGTH <= 1024); // You can assume the sequence length is not more than 1024

    const unsigned int numThreadsPerBlock = SEQUENCE_LENGTH;
    const unsigned int numBlocks = numSequences;
    //Launching the kernel
    hipDeviceSynchronize();
    kernel_nw2 <<< numBlocks, numThreadsPerBlock >>> (sequence1_d, sequence2_d, scores_d, numSequences);

}

