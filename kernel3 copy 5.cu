#include "hip/hip_runtime.h"

#include <assert.h>

#include "common.h"
#include "timer.h"

#define COARSEN_FACTOR 1

__global__ void kernel_nw3(unsigned char* sequence1, unsigned char* sequence2, int* scores_d, unsigned int numSequences)
{
    __shared__ int previousDiagonal[SEQUENCE_LENGTH / COARSEN_FACTOR];
    // __shared__ int sm_sequence2[SEQUENCE_LENGTH];    
    __shared__ int startLeft;
    __shared__ int currentIteration;
    startLeft = INSERTION * 1;

    if(threadIdx.x == 0) {
        currentIteration = 0;
    }
    // for (unsigned int factorIndex = 0; factorIndex < COARSEN_FACTOR; ++factorIndex) {
    //     int i = blockDim.x*factorIndex + threadIdx.x;
    //     sm_sequence2[i] = sequence2[blockIdx.x*SEQUENCE_LENGTH + i];
    // }
    // __syncthreads();

    int top,
        topLeft,
        left;
    int count = 2 * SEQUENCE_LENGTH - 1;

    // #pragma unroll
    for (unsigned int factorIndex = 0; factorIndex < COARSEN_FACTOR; ++factorIndex) {
        // __syncthreads();
        int row = 0;
        int col = threadIdx.x + blockDim.x*factorIndex;
        top     = (col + 1) * DELETION;
        topLeft = (col) * DELETION;
        int seq1Value = sequence1[blockIdx.x*SEQUENCE_LENGTH + col]; 

        // busy waiting other thread loop
        while(threadIdx.x < currentIteration);

        for (unsigned int diagIndex = 0 ; diagIndex <= count ; ++diagIndex) {
            // Compute current diagonal from left to right bottom to top
            
            if (threadIdx.x < diagIndex && row < SEQUENCE_LENGTH && col < SEQUENCE_LENGTH) {
                if(threadIdx.x == 0) {
                    ++currentIteration;
                }

                left    = (threadIdx.x == 0) ? startLeft : (previousDiagonal[threadIdx.x-1]);
                int insertion = top + INSERTION;
                int deletion  = left + DELETION;
                int match     = topLeft + (
                    (sequence2[blockIdx.x*SEQUENCE_LENGTH + (row)] == seq1Value)
                    // (sm_sequence2[(row-1)] == seq1Value)
                    ? MATCH 
                    : MISMATCH
                    ); //check if there is a match
                top = (insertion > deletion) ? insertion : deletion; 
                top = (match > top) ? match : top;
                //
                topLeft = left;
                ++row;
            }
            __syncthreads(); 

            previousDiagonal[threadIdx.x] = top;
            // __syncthreads();
        }

        if(row == 1 && threadIdx.x == blockDim.x - 1) {
            startLeft = top;
            printf("startLeft = top; %d\n", startLeft);
        }
        // left = 
    }
    // __syncthreads();
    // printf("top: %d\n", top);

    if(threadIdx.x == blockDim.x - 1){
        //printf("-- %d -- ** %d ** \n",previousDiagonal[SEQUENCE_LENGTH-1],previousDiagonal[SEQUENCE_LENGTH-2]);
        // 3 - Write the final score to the output array
        scores_d[blockIdx.x] = top;
    }
}


void nw_gpu3(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {

    assert(SEQUENCE_LENGTH <= 1024); // You can assume the sequence length is not more than 1024

    const unsigned int numThreadsPerBlock = SEQUENCE_LENGTH / COARSEN_FACTOR;
    const unsigned int numBlocks = numSequences;
    //Launching the kernel
    hipDeviceSynchronize();
    kernel_nw3 <<< numBlocks, numThreadsPerBlock >>> (sequence1_d, sequence2_d, scores_d, numSequences);

}
